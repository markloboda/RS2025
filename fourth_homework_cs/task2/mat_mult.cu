
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <stdio.h>
#define N 4096  
#define BLOCKSIZE 16
#define CEIL_DIV(M, N) (((M) + (N)-1) / (N))
#define WARP_SIZE 32

void init_matrices(half *a, half *b, float *c, int matsize) {
    for (int i = 0; i < matsize; ++i) {
        for (int j = 0; j < matsize; ++j) {
            a[i * matsize + j] = __float2half(1.0);
            b[i * matsize + j] = __float2half(1.0);
            c[i * matsize + j] = 1.0;
        }
    }
}


__global__ void mm_block_tc(int mat_size, half *A, half *B, float *C) {
    // Tile using a 2D grid
    int warpX = (blockIdx.x * blockDim.x + threadIdx.x) / WARP_SIZE;
    int warpY = (blockIdx.y * blockDim.y + threadIdx.y);
    //printf("warpM = %d, warpN = %d\n", warpX, warpY);
    int block_size = mat_size ;
    int aRow, aCol, bRow, bCol, cRow, cCol, dRow, dCol;
   
    // number of tiles 
    int num_tiles = CEIL_DIV(mat_size, BLOCKSIZE);
    int threadCol = threadIdx.x % BLOCKSIZE;
    int threadRow = threadIdx.x / BLOCKSIZE;

    // load A, B, C    
    half* A_block;
    half* B_block;
    float* C_block;

    // Continue
    
}


__global__ void mm_naive(int mat_size,  half *A, half *B, float *C) {
  // compute position in C that this thread is responsible for
  const uint x = blockIdx.x * blockDim.x + threadIdx.x;
  const uint y = blockIdx.y * blockDim.y + threadIdx.y;

  // `if` condition is necessary for when M or N aren't multiples of 32.
  if (x < mat_size && y < mat_size) {
    float tmp = 0.0;
    for (int i = 0; i < mat_size; ++i) {
      tmp += (float)A[x * mat_size + i] * (float)B[i * mat_size + y];
    }

    C[x * mat_size + y] = tmp ;
  }
}


int main() {
    half *mat_a, *mat_b;
    float *mat_c;

    mat_a = (half*)malloc(N * N * sizeof(half));
    mat_b = (half*)malloc(N * N * sizeof(half));
    mat_c = (float*)malloc(N * N * sizeof(float));

    // init matrices 
    init_matrices(mat_a, mat_b, mat_c, N);

    // instantiate buffers on the device
    half *d_mat_a, *d_mat_b;
    float *d_mat_c;
    hipMalloc(&d_mat_a, N * N * sizeof(half));
    hipMalloc(&d_mat_b, N * N * sizeof(half));
    hipMalloc(&d_mat_c, N * N * sizeof(float));

    // copy data from host to device
    hipMemcpy(d_mat_a, mat_a, N * N * sizeof(half), hipMemcpyHostToDevice);
    hipMemcpy(d_mat_b, mat_b, N * N * sizeof(half), hipMemcpyHostToDevice);
   
     // First: using WMM
    dim3 gridDim;
    dim3 blockDim; 
    // launch kernel
    blockDim.x = 32;
    blockDim.y = 1;
    gridDim.x = CEIL_DIV(N, BLOCKSIZE * blockDim.x / 32); 
    gridDim.y = CEIL_DIV(N, BLOCKSIZE * blockDim.y);

    hipEvent_t start1, stop1, start2, stop2;
    float milliseconds = 0;

    // to avoid CUDA compile overhead
    mm_naive<<<gridDim, blockDim>>>(N, d_mat_a, d_mat_b, d_mat_c);

    hipEventCreate(&start1);
    hipEventCreate(&stop1);
    hipEventCreate(&start2);
    hipEventCreate(&stop2);

    printf("Running gemm with Tensor cores...\n");
    hipEventRecord(start2);
    // Uncommment here to use TC
    //mm_block_tc<<<gridDim, blockDim>>>(N, d_mat_a, d_mat_b, d_mat_c);
    hipEventRecord(stop2);
    hipEventSynchronize(stop2);
    hipEventElapsedTime(&milliseconds, start2, stop2);
    printf("Time taken with TC: %f ms\n", milliseconds);


    printf("Running gemm without Tensor cores...\n");
    hipEventRecord(start1);
    mm_naive<<<gridDim, blockDim>>>(N, d_mat_a, d_mat_b, d_mat_c);
    hipEventRecord(stop1);
    hipEventSynchronize(stop1);
    hipEventElapsedTime(&milliseconds, start1, stop1);
    printf("Time taken without TC: %f ms\n", milliseconds);
    
    hipMemcpy(mat_c, d_mat_c, N * N * sizeof(float), hipMemcpyDeviceToHost);  

    // free memory
    free(mat_a);
    free(mat_b);
    free(mat_c);
    hipFree(d_mat_a);
    hipFree(d_mat_b);
    hipFree(d_mat_c);
    
    return 0;
}